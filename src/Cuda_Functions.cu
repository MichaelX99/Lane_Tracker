#include "hip/hip_runtime.h"
#include <Cuda_Functions.h>

const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

static void HandleCUDAError( hipError_t err,
                             const char *file,
                             int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_CUDA_ERROR( err ) (HandleCUDAError( err, __FILE__, __LINE__ ))

static void HandleCUBLASError( hipblasStatus_t err,
                               const char *file,
                               int line ) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        printf( "%s in %s at line %d\n", cublasGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_CUBLAS_ERROR( err ) (HandleCUBLASError( err, __FILE__, __LINE__ ))

const char* curandGetErrorString(hiprandStatus_t status)
{
    switch(status)
    {
        case HIPRAND_STATUS_SUCCESS: return "HIPRAND_STATUS_SUCCESS";
        case HIPRAND_STATUS_VERSION_MISMATCH: return "HIPRAND_STATUS_VERSION_MISMATCH";
        case HIPRAND_STATUS_NOT_INITIALIZED: return "HIPRAND_STATUS_NOT_INITIALIZED";
        case HIPRAND_STATUS_ALLOCATION_FAILED: return "HIPRAND_STATUS_ALLOCATION_FAILED";
        case HIPRAND_STATUS_TYPE_ERROR: return "HIPRAND_STATUS_TYPE_ERROR";
        case HIPRAND_STATUS_OUT_OF_RANGE: return "HIPRAND_STATUS_OUT_OF_RANGE";
        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE: return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED: return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
        case HIPRAND_STATUS_LAUNCH_FAILURE: return "HIPRAND_STATUS_LAUNCH_FAILURE";
        case HIPRAND_STATUS_PREEXISTING_FAILURE: return "HIPRAND_STATUS_PREEXISTING_FAILURE";
        case HIPRAND_STATUS_INITIALIZATION_FAILED: return "HIPRAND_STATUS_INITIALIZATION_FAILED";
        case HIPRAND_STATUS_ARCH_MISMATCH: return "HIPRAND_STATUS_ARCH_MISMATCH";
        case HIPRAND_STATUS_INTERNAL_ERROR: return "HIPRAND_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

static void HandleCURANDError( hiprandStatus_t err,
                               const char *file,
                               int line ) {
    if (err != HIPRAND_STATUS_SUCCESS) {
        printf( "%s in %s at line %d\n", curandGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_CURAND_ERROR( err ) (HandleCURANDError( err, __FILE__, __LINE__ ))

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void normalize(float* d_particle_matrix, const int num_states, const int num_particles)
{

}

__global__ void average_matrix(float* d_particle_matrix, float* d_avg_particle, const int num_states, const int num_particles)
{

}

float* cuda_fill_rand(float *d_particle_matrix, const int num_states, const int num_particles)
{
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	HANDLE_CURAND_ERROR( hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT) );

	// Set the seed for the random number generator using the system clock
	HANDLE_CURAND_ERROR( hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock()) );

	// Fill the array with random numbers on the device
	HANDLE_CURAND_ERROR( hiprandGenerateUniform(prng, d_particle_matrix, num_states * num_particles) );

  return d_particle_matrix;
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
float* gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n, hipblasHandle_t handle)
{
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	HANDLE_CUBLAS_ERROR( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc) );

  return C;
}

float* cuda_normalize_particles(float *d_particle_matrix, const int num_states, const int num_particles)
{
  normalize<<<1, 1>>> (d_particle_matrix, num_states, num_particles);

  return d_particle_matrix;
}

float* cuda_initialize_particles(float* d_particles, const int num_states, const int num_particles)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&d_particles, num_states * num_particles * sizeof(float)) );
  d_particles = cuda_fill_rand(d_particles, num_particles, num_states);
  d_particles = cuda_normalize_particles(d_particles, num_particles, num_states);

  return d_particles;
}

float* cuda_copy_transition_matrix(float* h_transition_matrix, float* d_transition_matrix, const int num_states)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&d_transition_matrix, num_states * num_states * sizeof(float)) );
  HANDLE_CUDA_ERROR( hipMemcpy(d_transition_matrix, h_transition_matrix, num_states * num_states * sizeof(float), hipMemcpyHostToDevice) );

  return d_transition_matrix;
}

int cuda_compute_argmax_state(hipblasHandle_t handle,float* d_particle_matrix, float* d_avg_particle, const int num_states, const int num_particles)
{
  int state;

  // compute the average particle state
  average_matrix<<<1, 1>>> (d_particle_matrix, d_avg_particle, num_states, num_particles);

  // find the argmax
  HANDLE_CUBLAS_ERROR( hipblasIsamax(handle, num_states, d_avg_particle, 1, &state) );

  return state;
}

float* cuda_apply_transition(hipblasHandle_t handle, float* particles, float* transition, const int num_states, const int num_particles)
{
  // matrix multiply
  particles = gpu_blas_mmul(particles, particles, transition, num_particles, num_states, num_states, handle);

  // normalize matrix
  particles = cuda_normalize_particles(particles, num_states, num_particles);

  return particles;
}

float* initialize_gpu_array(float* A, const int num_states)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&A, num_states * sizeof(float)) );

  return A;
}

float* cuda_form_obs_vector(float* sensor_observation, const int index)
{

  return sensor_observation;
}

float* cuda_reweight_particles(float* particle_matrix, float* sensor_observation, const int num_states, const int num_particles)
{
  // write element wise multiplcation kernel

  return particle_matrix;
}

float* cuda_resample_particles(float* particle_matrix)
{

  return particle_matrix;
}

void cuda_destroy(float* d_A)
{
  hipFree(d_A);
}
