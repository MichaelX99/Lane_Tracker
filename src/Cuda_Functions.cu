#include "hip/hip_runtime.h"
#include <Cuda_Functions.h>

const int BLOCK_SIZE = 16;

const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

static void HandleCUDAError( hipError_t err,
                             const char *file,
                             int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_CUDA_ERROR( err ) (HandleCUDAError( err, __FILE__, __LINE__ ))

static void HandleCUBLASError( hipblasStatus_t err,
                               const char *file,
                               int line ) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        printf( "%s in %s at line %d\n", cublasGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_CUBLAS_ERROR( err ) (HandleCUBLASError( err, __FILE__, __LINE__ ))

const char* curandGetErrorString(hiprandStatus_t status)
{
    switch(status)
    {
        case HIPRAND_STATUS_SUCCESS: return "HIPRAND_STATUS_SUCCESS";
        case HIPRAND_STATUS_VERSION_MISMATCH: return "HIPRAND_STATUS_VERSION_MISMATCH";
        case HIPRAND_STATUS_NOT_INITIALIZED: return "HIPRAND_STATUS_NOT_INITIALIZED";
        case HIPRAND_STATUS_ALLOCATION_FAILED: return "HIPRAND_STATUS_ALLOCATION_FAILED";
        case HIPRAND_STATUS_TYPE_ERROR: return "HIPRAND_STATUS_TYPE_ERROR";
        case HIPRAND_STATUS_OUT_OF_RANGE: return "HIPRAND_STATUS_OUT_OF_RANGE";
        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE: return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED: return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
        case HIPRAND_STATUS_LAUNCH_FAILURE: return "HIPRAND_STATUS_LAUNCH_FAILURE";
        case HIPRAND_STATUS_PREEXISTING_FAILURE: return "HIPRAND_STATUS_PREEXISTING_FAILURE";
        case HIPRAND_STATUS_INITIALIZATION_FAILED: return "HIPRAND_STATUS_INITIALIZATION_FAILED";
        case HIPRAND_STATUS_ARCH_MISMATCH: return "HIPRAND_STATUS_ARCH_MISMATCH";
        case HIPRAND_STATUS_INTERNAL_ERROR: return "HIPRAND_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

static void HandleCURANDError( hiprandStatus_t err,
                               const char *file,
                               int line ) {
    if (err != HIPRAND_STATUS_SUCCESS) {
        printf( "%s in %s at line %d\n", curandGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_CURAND_ERROR( err ) (HandleCURANDError( err, __FILE__, __LINE__ ))

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void normalize(float* d_particle_matrix, float* d_row_sum, const int num_states, const int num_particles)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int matrix_ind;
  int vector_ind;

  if (row < num_particles && col < num_states)
  {
    matrix_ind = num_states * row + col;
    vector_ind = row;
    d_particle_matrix[matrix_ind] /= d_row_sum[vector_ind];
  }
}

__global__ void average_matrix(float* d_particle_matrix, float* d_avg_particle, const int num_states, const int num_particles)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < num_states && col < num_particles)
  {

  }
}

void print_matrix(char* name, float* A, const int rows, const int cols)
{
  float* h_A = (float*)malloc(rows * cols * sizeof(float));

  HANDLE_CUDA_ERROR( hipMemcpy(h_A, A, rows * cols * sizeof(float), hipMemcpyDeviceToHost) );

  int index;

  printf(name);
  printf("\n");

  for (int i = 0; i < rows; i++)
  {
    for (int j = 0; j < cols; j++)
    {
      index = i * cols + j;
      printf("%f ", h_A[index]);
    }
    printf("\n");
  }
  printf("\n\n");

  free(h_A);
}

float* cuda_fill_rand(float *d_particle_matrix, const int num_states, const int num_particles)
{
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	HANDLE_CURAND_ERROR( hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT) );

	// Set the seed for the random number generator using the system clock
	HANDLE_CURAND_ERROR( hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock()) );

	// Fill the array with random numbers on the device
	HANDLE_CURAND_ERROR( hiprandGenerateUniform(prng, d_particle_matrix, num_states * num_particles) );

  return d_particle_matrix;
}

float* initialize_gpu_ones(float* A, const int size)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&A, size * sizeof(float)) );
  float* h_A = (float*) malloc(size * sizeof(float));
  for (int i = 0; i < size; i++)
  {
    h_A[i] = 1.0;
  }
  HANDLE_CUDA_ERROR( hipMemcpy(A, h_A, size * sizeof(float), hipMemcpyHostToDevice) );

  free(h_A);

  return A;
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
float* gpu_blas_mmul(float *A, float *B, float *C, const int m, const int k, const int n, hipblasHandle_t handle)
{
	//int lda=m,ldb=k,ldc=m;
  int lda=m,ldb=m,ldc=n;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	//HANDLE_CUBLAS_ERROR( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc) );

  HANDLE_CUBLAS_ERROR( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, m, B, k, beta, C, m) );

  return C;
}

// Matrix Vector Multiplication
// c(m,1) = A(m,n) * b(n,1)
float* gpu_blas_vmul(char* type, const float *A, const float *b, float *c, const int m, const int n, hipblasHandle_t handle)
{
  int lda=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

  if (strcmp(type, "row") == 0)
  {
    HANDLE_CUBLAS_ERROR( hipblasSgemv(handle, HIPBLAS_OP_T, m, n, alpha, A, lda, b, 1, beta, c, 1) );
  }
  else if (strcmp(type, "col") == 0)
  {
    HANDLE_CUBLAS_ERROR( hipblasSgemv(handle, HIPBLAS_OP_N, m, n, alpha, A, lda, b, 1, beta, c, 1) );
  }


  return c;
}

float* cuda_normalize_particles(float *d_particle_matrix, float* d_particle_ones, float* d_row_sum, const int num_states, const int num_particles, hipblasHandle_t handle)
{
  d_row_sum = gpu_blas_vmul("row", d_particle_matrix, d_particle_ones, d_row_sum, num_states, num_particles, handle);

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((num_states + dimBlock.x - 1) / dimBlock.x,
               (num_particles + dimBlock.y - 1) / dimBlock.y);

  normalize<<<dimGrid, dimBlock>>> (d_particle_matrix, d_row_sum, num_states, num_particles);

  HANDLE_CUDA_ERROR( hipDeviceSynchronize() );

  return d_particle_matrix;
}

float* cuda_initialize_particles(hipblasHandle_t handle, float* d_particles, float* d_ones, float* d_row_sum, const int num_states, const int num_particles)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&d_particles, num_states * num_particles * sizeof(float)) );
  //d_particles = cuda_fill_rand(d_particles, num_particles, num_states);
  float* particles = (float*)malloc(num_states * num_particles * sizeof(float));

  float val;
  for (int i = 0; i < num_states*num_particles; i++)
  {
    particles[i] = i;
  }

  HANDLE_CUDA_ERROR( hipMemcpy(d_particles, particles, num_particles * num_states * sizeof(float), hipMemcpyHostToDevice) );
  free(particles);


  d_particles = cuda_normalize_particles(d_particles, d_ones, d_row_sum, num_states, num_particles, handle);

  return d_particles;
}

float* cuda_copy_transition_matrix(float* h_transition_matrix, float* d_transition_matrix, const int num_states)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&d_transition_matrix, num_states * num_states * sizeof(float)) );
  HANDLE_CUDA_ERROR( hipMemcpy(d_transition_matrix, h_transition_matrix, num_states * num_states * sizeof(float), hipMemcpyHostToDevice) );

  return d_transition_matrix;
}

int cuda_compute_argmax_state(hipblasHandle_t handle,float* d_particle_matrix, float* d_avg_particle, const int num_states, const int num_particles)
{
  int state;

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((num_states + dimBlock.x - 1) / dimBlock.x,
               (num_particles + dimBlock.y - 1) / dimBlock.y);

  // compute the average particle state
  average_matrix<<<dimGrid, dimBlock>>> (d_particle_matrix, d_avg_particle, num_states, num_particles);

  HANDLE_CUDA_ERROR( hipDeviceSynchronize() );

  // find the argmax
  HANDLE_CUBLAS_ERROR( hipblasIsamax(handle, num_states, d_avg_particle, 1, &state) );

  return state;
}

float* cuda_apply_transition(hipblasHandle_t handle, float* particles, float* transition, float* d_ones, float* d_sum, const int num_states, const int num_particles)
{
  // matrix multiply
  particles = gpu_blas_mmul(transition, particles, particles, num_states, num_states, num_particles, handle);

  // normalize matrix
  particles = cuda_normalize_particles(particles, d_ones, d_sum, num_states, num_particles, handle);

  return particles;
}

float* initialize_gpu_array(float* A, const int num_states)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&A, num_states * sizeof(float)) );

  return A;
}

float* cuda_form_obs_vector(float* sensor_observation, const int index)
{

  return sensor_observation;
}

float* cuda_reweight_particles(float* particle_matrix, float* sensor_observation, const int num_states, const int num_particles)
{
  // write element wise multiplcation kernel

  return particle_matrix;
}

float* cuda_resample_particles(float* particle_matrix)
{

  return particle_matrix;
}

void cuda_destroy(float* d_A)
{
  hipFree(d_A);
}
