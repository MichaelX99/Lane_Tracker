#include "hip/hip_runtime.h"
#include <Cuda_Functions.h>

const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "unknown error";
}

static void HandleCUDAError( hipError_t err,
                             const char *file,
                             int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_CUDA_ERROR( err ) (HandleCUDAError( err, __FILE__, __LINE__ ))

static void HandleCUBLASError( hipblasStatus_t err,
                               const char *file,
                               int line ) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        printf( "%s in %s at line %d\n", cublasGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_CUBLAS_ERROR( err ) (HandleCUBLASError( err, __FILE__, __LINE__ ))

void cuda_fill_rand(float *A, int rows, int cols)
{
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	hiprandGenerateUniform(prng, A, rows * cols);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n, hipblasHandle_t handle)
{
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	HANDLE_CUBLAS_ERROR( hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc) );
}

void cuda_normalize_particles(float *A, int rows, int cols)
{

}

void cuda_initialize_particles(float* d_particles, const int num_states, const int num_particles)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&d_particles, num_states * num_particles * sizeof(float)) );
  cuda_fill_rand(d_particles, num_particles, num_states);
  cuda_normalize_particles(d_particles, num_particles, num_states);
}

void cuda_copy_transition_matrix(float* h_transition_matrix, float* d_transition_matrix, const int num_states)
{
  HANDLE_CUDA_ERROR( hipMalloc((void**)&d_transition_matrix, num_states * num_states * sizeof(float)) );
  HANDLE_CUDA_ERROR( hipMemcpy(d_transition_matrix, h_transition_matrix, num_states * num_states * sizeof(float), hipMemcpyHostToDevice) );
}

int cuda_compute_argmax_state(float* d_particles, const int num_states, const int num_particles)
{
  int state = 0;

  return state;
}

void cuda_apply_transition(hipblasHandle_t handle)
{
  // matrix multiply

  // normalize matrix
}

void cuda_form_obs_vector()
{

}

void cuda_reweight_particles()
{

}

void cuda_resample_particles()
{

}

void cuda_destroy(float* d_A)
{
  hipFree(d_A);
}
