// Low level matrix multiplication on GPU using CUDA with CURAND and CUBLAS
// C(m,n) = A(m,k) * B(k,n)
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hipblas.h>
#include <hiprand/hiprand.h>
//#include <ctime>// include this header
#include <stdio.h>
#include <hip/hip_runtime.h>



// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	// Create a pseudo-random number generator
	hiprandGenerator_t prng;
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

	// Fill the array with random numbers on the device
	//curandGenerateUniform(prng, A, nr_rows_A * nr_cols_A);
}

// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const float *A, const float *B, float *C, const int m, const int k, const int n, hipblasHandle_t handle) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "CUBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "CUBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "CUBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "CUBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "CUBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "CUBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "CUBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "CUBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}

//Print matrix A(nr_rows_A, nr_cols_A) storage in column-major format
void print_matrix(const float *A, int nr_rows_A, int nr_cols_A) {

    for(int i = 0; i < nr_rows_A; ++i){
        for(int j = 0; j < nr_cols_A; ++j){
            std::cout << A[j * nr_rows_A + i] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
	hipblasStatus_t stat;

	// Allocate 3 arrays on CPU
	int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;

  int num_particles = 5000;
  int num_states = 3000;
  nr_rows_A = nr_rows_C = num_particles;
  nr_cols_A = nr_rows_B = num_states;
  nr_cols_B = nr_cols_C = num_states;

	float *h_A = (float *)malloc(nr_rows_A * nr_cols_A * sizeof(float));
	float *h_B = (float *)malloc(nr_rows_B * nr_cols_B * sizeof(float));
	float *h_C = (float *)malloc(nr_rows_C * nr_cols_C * sizeof(float));

	// Allocate 3 arrays on GPU
	float *d_A, *d_B, *d_C;
	hipMalloc((void**)&d_A,nr_rows_A * nr_cols_A * sizeof(float));
	hipMalloc((void**)&d_B,nr_rows_B * nr_cols_B * sizeof(float));
	hipMalloc((void**)&d_C,nr_rows_C * nr_cols_C * sizeof(float));

	// Fill the arrays A and B on GPU with random numbers
	GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
	GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  stat = hipblasCreate(&handle);

	std::cout << _cudaGetErrorEnum(stat) << "\n";

	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		printf("error\n");
		return;
	}

  int start_s = clock();
	// Multiply A and B on GPU
	gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B, handle);

  //cudaMemcpy(h_C,d_C,nr_rows_C * nr_cols_C * sizeof(float),cudaMemcpyDeviceToHost);
	int stop_s = clock();
  std::cout << "time: " << (stop_s-start_s)/double(CLOCKS_PER_SEC) << std::endl;


  // Destroy the handle
  hipblasDestroy(handle);

	//Free GPU memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Free CPU memory
	free(h_A);
	free(h_B);
	free(h_C);

	return 0;
}
